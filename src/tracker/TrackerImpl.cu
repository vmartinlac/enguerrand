#include "hip/hip_runtime.h"
#include <queue>
#include <opencv2/core/cuda.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include "TrackerImpl.h"

__global__ void mykernel(cv::cuda::PtrStep<float> tmp)
{
}

TrackerImpl::TrackerImpl()
{
    mNeighbors[0] = cv::Vec2i(-1,-1);
    mNeighbors[1] = cv::Vec2i(-1,0);
    mNeighbors[2] = cv::Vec2i(-1,1);
    mNeighbors[3] = cv::Vec2i(0,-1);
    mNeighbors[4] = cv::Vec2i(0,1);
    mNeighbors[5] = cv::Vec2i(1,-1);
    mNeighbors[6] = cv::Vec2i(1,0);
    mNeighbors[7] = cv::Vec2i(1,1);
}

void TrackerImpl::track(const cv::Mat& input_image, std::vector<TrackedLandmark>& result)
{
    detectEdges(input_image);
    findCircles();
}

void TrackerImpl::findCircles()
{
    const cv::Size image_size = mFlags.size();

    std::vector<cv::Vec2i> pixels_to_process;
    
    pixels_to_process.reserve(image_size.width*image_size.height);

    for(int i=0; i<image_size.height; i++)
    {
        for(int j=0; j<image_size.width; j++)
        {
            if(mFlags(i,j) & FLAG_EDGE)
            {
                pixels_to_process.push_back(cv::Vec2i(i,j));
            }
        }
    }

    while(pixels_to_process.empty() == false)
    {
        std::uniform_int_distribution<int> distrib(0, pixels_to_process.size()-1);

        const int selected_index = distrib(mEngine);

        const cv::Vec2i seed = pixels_to_process[selected_index];
        pixels_to_process[selected_index] = pixels_to_process.back();
        pixels_to_process.pop_back();

        const int i = seed[0];
        const int j = seed[1];

        if( (mFlags(i,j) & FLAG_NO_SEED) == 0 )
        {
            findCircle(seed);
        }
    }
}

void TrackerImpl::findCircle(const cv::Vec2i& seed)
{
    std::vector<cv::Vec2i> patch;
    bool ok = true;

    auto pred_radius = [this,seed] (const cv::Vec2i& neighbor)
    {
        const float dist = std::hypot(
            float(seed[0]-neighbor[0]),
            float(seed[1]-neighbor[1]));

        const float radius = 6.0f;

        return (dist < radius);
    };

    auto pred_line = [this] (const cv::Vec2i& neighbor)
    {
        return false;
    };

    if(ok)
    {
        patch.push_back(seed);
        growPatch(patch, pred_radius);
        ok = (patch.size() >= 5);
    }

    if(ok)
    {
    }
}

template<typename T>
void TrackerImpl::growPatch<T>(std::vector<cv::Vec2i>& patch, const T& pred)
{
    std::vector<cv::Vec2i> visited;
    std::queue<cv::Vec2i> queue;

    for(cv::Vec2i& pt : patch)
    {
        mFlags(pt) |= FLAG_VISITED;
        visited.push_back(pt);
        queue.push(pt);
    }

    while( queue.empty() == false )
    {
        const cv::Vec2i point = queue.front();
        queue.pop();

        for(int k=0; k<mNeighbors.size(); k++)
        {
            const cv::Vec2i neighbor = point + mNeighbors[k];

            if( 0 <= neighbor[0] && neighbor[0] < mFlags.rows && 0 <= neighbor[1] && neighbor[1] < mFlags.cols )
            {
                const uint8_t f = mFlags(neighbor);

                if( (f & FLAG_EDGE) != 0 && (f & FLAG_VISITED) == 0 && pred(neighbor))
                {
                    mFlags(neighbor) |= FLAG_VISITED;
                    visited.push_back(neighbor);
                    patch.push_back(neighbor);
                }
            }
        }
    }

    for( const cv::Vec2i& pt : visited )
    {
        mFlags(pt) &= ~FLAG_VISITED;
    }
}

void TrackerImpl::detectEdges(const cv::Mat& input_image)
{
    if( input_image.type() != CV_8UC3 ) throw "Internal error: incorrect image format.";

    const cv::Size image_size = input_image.size();

    cv::Mat3f sobel_x;
    cv::Mat3f sobel_y;
    cv::Mat3f sobel_norm(image_size);
    cv::Mat1f max_sobel_norm(image_size);
    cv::Mat1b channel( image_size );

    mFlags.create( image_size );

    // compute sobel x-derivative and y-derivative.

    cv::Sobel(input_image, sobel_x, CV_32F, 1, 0, 5);
    cv::Sobel(input_image, sobel_y, CV_32F, 0, 1, 5);

    // compute norm of gradient.


    for(int i=0; i<image_size.height; i++)
    {
        for(int j=0; j<image_size.width; j++)
        {
            sobel_norm(i,j)[0] = std::hypot(sobel_x(i,j)[0], sobel_y(i,j)[0]);
            sobel_norm(i,j)[1] = std::hypot(sobel_x(i,j)[1], sobel_y(i,j)[1]);
            sobel_norm(i,j)[2] = std::hypot(sobel_x(i,j)[2], sobel_y(i,j)[2]);
            max_sobel_norm(i,j) = std::max(sobel_norm(i,j)[0], std::max( sobel_norm(i,j)[1], sobel_norm(i,j)[2] ));
        }
    }

    // for each pixel, compute the channel on which its gradient has the largest margnitude.

    for(int i=0; i<image_size.height; i++)
    {
        for(int j=0; j<image_size.width; j++)
        {
            const cv::Vec3f col = sobel_norm(i,j);

            int c = 0;
            float value = col[0];

            if( col[1] > value )
            {
                c = 1;
                value = col[1];
            }

            if( col[2] > value )
            {
                c = 2;
                value = col[2];
            }

            channel(i,j) = c;
        }
    }

    // non-maximum suppression.

    const int margin = 3;

    for(int i=0; i<image_size.height; i++)
    {
        for(int j=0; j<image_size.width; j++)
        {
            uint8_t flag = 0;

            if( margin <= i && i+margin < image_size.height && margin <= j && j+margin < image_size.width )
            {
                const int c = channel(i,j);
                const float gradient_x = sobel_x(i,j)[c];
                const float gradient_y = sobel_y(i,j)[c];
                const float gradient_norm = sobel_norm(i,j)[c];

                const double epsilon = 1.0e-5;

                if( gradient_norm > epsilon )
                {
                    int dx = 0;
                    int dy = 0;

                    if(gradient_x/gradient_norm <= -M_SQRT1_2)
                    {
                        dx = -1;
                    }
                    else if(gradient_x/gradient_norm >= M_SQRT1_2)
                    {
                        dx = 1;
                    }

                    if(gradient_y/gradient_norm <= -M_SQRT1_2)
                    {
                        dy = -1;
                    }
                    else if(gradient_y/gradient_norm >= M_SQRT1_2)
                    {
                        dy = 1;
                    }

                    if( dx != 0 || dy != 0 )
                    {
                        const bool ismax = ( gradient_norm >= sobel_norm(i+dy,j+dx)[c] ) && ( gradient_norm >= sobel_norm(i-dy,j-dx)[c] );

                        if( ismax )
                        {
                            flag = FLAG_EDGE;
                        }
                    }
                }
            }

            mFlags(i,j) = flag;
        }
    }

    //cv::imshow("rien", mFlags*255);
    //cv::waitKey(0);

    {
        // TODO: use dichotomy to compute quantiles.

        std::vector<float> values(image_size.width*image_size.height);
        std::copy(max_sobel_norm.begin(), max_sobel_norm.end(), values.begin());

        std::sort(values.begin(), values.end());

        int k1 = static_cast<int>( values.size()*0.97 );
        k1 = std::min<int>(values.size(), std::max(0,k1));
        const float high_threshold = values[k1];

        int k2 = static_cast<int>( values.size()*0.92 );
        k2 = std::min<int>(values.size(), std::max(0,k2));
        const float low_threshold = values[k2];

        for(int i=0; i<image_size.height; i++)
        {
            for(int j=0; j<image_size.width; j++)
            {
                if( mFlags(i,j) == FLAG_EDGE )
                {
                    const float this_value = max_sobel_norm(i,j);

                    if( this_value < high_threshold )
                    {
                        mFlags(i,j) = 0;

                        if( this_value >= low_threshold )
                        {
                            for(const cv::Vec2i& delta : mNeighbors)
                            {
                                const cv::Vec2i that_point(i+delta[0], j+delta[1]);

                                if( 0 <= that_point[0] && that_point[0] < image_size.height && 0 <= that_point[1] && that_point[1] < image_size.width )
                                {
                                    const float that_value = max_sobel_norm(that_point[0], that_point[1]);

                                    if(that_value > high_threshold)
                                    {
                                        mFlags(i,j) = FLAG_EDGE;
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }

    //cv::imshow("rien", mFlags*255);
    //cv::waitKey(0);
}

